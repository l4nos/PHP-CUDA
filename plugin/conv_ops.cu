#include "conv_ops.cuh"

extern "C" hipError_t cuda_batch_convolution_kernel(
    hipdnnHandle_t handle,
    const float* const input_arrays[],
    const float* const filter_arrays[],
    float* const output_arrays[],
    int batch_count,
    int batch_size,
    int in_channels,
    int height,
    int width,
    int filter_count,
    int filter_height,
    int filter_width,
    hipStream_t stream
) {
    hipdnnTensorDescriptor_t* input_descs = new hipdnnTensorDescriptor_t[batch_count];
    hipdnnTensorDescriptor_t* output_descs = new hipdnnTensorDescriptor_t[batch_count];
    hipdnnFilterDescriptor_t* filter_descs = new hipdnnFilterDescriptor_t[batch_count];
    hipdnnConvolutionDescriptor_t* conv_descs = new hipdnnConvolutionDescriptor_t[batch_count];
    
    // Initialize descriptors
    for (int i = 0; i < batch_count; i++) {
        hipdnnCreateTensorDescriptor(&input_descs[i]);
        hipdnnCreateTensorDescriptor(&output_descs[i]);
        hipdnnCreateFilterDescriptor(&filter_descs[i]);
        hipdnnCreateConvolutionDescriptor(&conv_descs[i]);
        
        hipdnnSetTensor4dDescriptor(
            input_descs[i],
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            batch_size,
            in_channels,
            height,
            width
        );
        
        hipdnnSetFilter4dDescriptor(
            filter_descs[i],
            HIPDNN_DATA_FLOAT,
            HIPDNN_TENSOR_NCHW,
            filter_count,
            in_channels,
            filter_height,
            filter_width
        );
        
        hipdnnSetConvolution2dDescriptor(
            conv_descs[i],
            1, 1, // padding
            1, 1, // stride
            1, 1, // dilation
            HIPDNN_CROSS_CORRELATION,
            HIPDNN_DATA_FLOAT
        );
        
        int out_n, out_c, out_h, out_w;
        hipdnnGetConvolution2dForwardOutputDim(
            conv_descs[i],
            input_descs[i],
            filter_descs[i],
            &out_n,
            &out_c,
            &out_h,
            &out_w
        );
        
        hipdnnSetTensor4dDescriptor(
            output_descs[i],
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            out_n,
            out_c,
            out_h,
            out_w
        );
    }
    
    // Find best algorithm
    hipdnnConvolutionFwdAlgo_t algo;
    hipdnnGetConvolutionForwardAlgorithm(
        handle,
        input_descs[0],
        filter_descs[0],
        conv_descs[0],
        output_descs[0],
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        0,
        &algo
    );
    
    // Get workspace size
    size_t workspace_size = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(
        handle,
        input_descs[0],
        filter_descs[0],
        conv_descs[0],
        output_descs[0],
        algo,
        &workspace_size
    );
    
    // Allocate workspace
    void* workspace = nullptr;
    if (workspace_size > 0) {
        hipMalloc(&workspace, workspace_size);
    }
    
    // Perform batch convolution
    float alpha = 1.0f;
    float beta = 0.0f;
    
    for (int i = 0; i < batch_count; i++) {
        hipdnnStatus_t status = hipdnnConvolutionForward(
            handle,
            &alpha,
            input_descs[i],
            input_arrays[i],
            filter_descs[i],
            filter_arrays[i],
            conv_descs[i],
            algo,
            workspace,
            workspace_size,
            &beta,
            output_descs[i],
            output_arrays[i]
        );
        
        if (status != HIPDNN_STATUS_SUCCESS) {
            // Cleanup
            if (workspace) hipFree(workspace);
            for (int j = 0; j < batch_count; j++) {
                hipdnnDestroyTensorDescriptor(input_descs[j]);
                hipdnnDestroyTensorDescriptor(output_descs[j]);
                hipdnnDestroyFilterDescriptor(filter_descs[j]);
                hipdnnDestroyConvolutionDescriptor(conv_descs[j]);
            }
            delete[] input_descs;
            delete[] output_descs;
            delete[] filter_descs;
            delete[] conv_descs;
            return hipErrorUnknown;
        }
    }
    
    // Cleanup
    if (workspace) hipFree(workspace);
    for (int i = 0; i < batch_count; i++) {
        hipdnnDestroyTensorDescriptor(input_descs[i]);
        hipdnnDestroyTensorDescriptor(output_descs[i]);
        hipdnnDestroyFilterDescriptor(filter_descs[i]);
        hipdnnDestroyConvolutionDescriptor(conv_descs[i]);
    }
    delete[] input_descs;
    delete[] output_descs;
    delete[] filter_descs;
    delete[] conv_descs;
    
    if (stream == 0) {
        return hipDeviceSynchronize();
    }
    
    return hipSuccess;
}
