#include "tensor_core_ops.cuh"
#include <hip/hip_fp16.h>

// Helper function to check Tensor Core support
bool check_tensor_core_support(hipDeviceProp_t& prop) {
    return prop.major >= 7;  // Volta or newer
}

extern "C" hipError_t cuda_tensorcore_matmul(
    hipblasHandle_t handle,
    const void* A,
    const void* B,
    void* C,
    int m, int n, int k,
    TensorCoreConfig* config
) {
    if (!config->enabled) {
        return hipErrorNotSupported;
    }

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasStatus_t status = hipblasGemmEx(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m, n, k,
        &alpha,
        A, config->input_type, m,
        B, config->input_type, k,
        &beta,
        C, config->output_type, m,
        config->compute_type,
        config->algo
    );

    return (status == HIPBLAS_STATUS_SUCCESS) ? hipSuccess : hipErrorUnknown;
}

extern "C" hipError_t cuda_mixed_precision_matmul(
    hipblasHandle_t handle,
    const half* A,
    const half* B,
    float* C,
    int m, int n, int k
) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasStatus_t status = hipblasGemmEx(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m, n, k,
        &alpha,
        A, HIP_R_16F, m,
        B, HIP_R_16F, k,
        &beta,
        C, HIP_R_32F, m,
        HIPBLAS_COMPUTE_32F_FAST_16F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );

    return (status == HIPBLAS_STATUS_SUCCESS) ? hipSuccess : hipErrorUnknown;
}

extern "C" hipError_t cuda_tensorcore_autotune(
    int m, int n, int k,
    TensorCoreConfig* config
) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    if (!check_tensor_core_support(prop)) {
        config->enabled = false;
        return hipSuccess;
    }

    // Auto-select precision based on size
    if (m * n * k > 1024 * 1024) {  // Large matrix
        config->input_type = HIP_R_16F;
        config->output_type = HIP_R_32F;
        config->compute_type = HIPBLAS_COMPUTE_32F_FAST_16F;
    } else {  // Small matrix
        config->input_type = HIP_R_32F;
        config->output_type = HIP_R_32F;
        config->compute_type = HIPBLAS_COMPUTE_32F;
    }

    // Select algorithm based on matrix size
    if (m % 8 == 0 && n % 8 == 0 && k % 8 == 0) {
        config->algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
    } else {
        config->algo = HIPBLAS_GEMM_DEFAULT;
    }

    config->enabled = true;
    return hipSuccess;
}
