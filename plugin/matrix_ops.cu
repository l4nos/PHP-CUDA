#include "matrix_ops.cuh"

extern "C" hipError_t cuda_batch_matrix_multiply_kernel(
    hipblasHandle_t handle,
    const float* const array_a[],
    const float* const array_b[],
    float* const array_c[],
    int batch_size,
    int m, int n, int k,
    hipStream_t stream
) {
    const float alpha = 1.0f;
    const float beta = 0.0f;
    
    hipblasStatus_t status = hipblasSgemmBatched(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        k, m, n,
        &alpha,
        array_b, k,
        array_a, n,
        &beta,
        array_c, k,
        batch_size
    );
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return hipErrorUnknown;
    }
    
    if (stream == 0) {
        return hipDeviceSynchronize();
    }
    
    return hipSuccess;
}
