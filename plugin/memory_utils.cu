#include "memory_utils.cuh"

extern "C" hipError_t cuda_pinned_malloc(void** ptr, size_t size) {
    return hipHostAlloc(ptr, size, hipHostMallocDefault);
}

extern "C" hipError_t cuda_pinned_free(void* ptr) {
    return hipHostFree(ptr);
}

extern "C" hipError_t cuda_unified_malloc(
    void** ptr,
    size_t size,
    MemoryConfig* config
) {
    hipError_t err = hipMallocManaged(ptr, size);
    if (err != hipSuccess) return err;

    // Apply memory hints
    if (config->read_mostly) {
        err = hipMemAdvise(*ptr, size, hipMemAdviseSetReadMostly, 0);
        if (err != hipSuccess) return err;
    }

    if (config->preferred_location >= 0) {
        err = hipMemAdvise(*ptr, size, hipMemAdviseSetPreferredLocation, 
                           config->preferred_location);
        if (err != hipSuccess) return err;
    }

    return hipSuccess;
}

extern "C" hipError_t cuda_unified_free(void* ptr) {
    return hipFree(ptr);
}

extern "C" hipError_t cuda_unified_prefetch(
    void* ptr,
    size_t size,
    int device
) {
    hipStream_t stream;
    hipError_t err = hipStreamCreate(&stream);
    if (err != hipSuccess) return err;

    err = hipMemPrefetchAsync(ptr, size, device, stream);
    if (err != hipSuccess) {
        hipStreamDestroy(stream);
        return err;
    }

    err = hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    return err;
}

extern "C" hipError_t cuda_set_memory_hint(
    void* ptr,
    size_t size,
    hipMemoryAdvise advice
) {
    return hipMemAdvise(ptr, size, advice, 0);
}

extern "C" hipError_t cuda_optimize_memory_access(
    void* ptr,
    size_t size,
    int device
) {
    // First, set preferred location
    hipError_t err = hipMemAdvise(ptr, size, 
                                   hipMemAdviseSetPreferredLocation, device);
    if (err != hipSuccess) return err;

    // Then, set accessed-by hint
    err = hipMemAdvise(ptr, size, hipMemAdviseSetAccessedBy, device);
    if (err != hipSuccess) return err;

    // Finally, prefetch to device
    return cuda_unified_prefetch(ptr, size, device);
}

extern "C" hipError_t cuda_measure_memory_bandwidth(
    size_t size,
    float* bandwidth
) {
    void *d_a, *d_b;
    hipEvent_t start, stop;
    float elapsed_time;

    // Allocate memory
    hipError_t err = hipMalloc(&d_a, size);
    if (err != hipSuccess) return err;

    err = hipMalloc(&d_b, size);
    if (err != hipSuccess) {
        hipFree(d_a);
        return err;
    }

    // Create events
    err = hipEventCreate(&start);
    if (err != hipSuccess) {
        hipFree(d_a);
        hipFree(d_b);
        return err;
    }

    err = hipEventCreate(&stop);
    if (err != hipSuccess) {
        hipEventDestroy(start);
        hipFree(d_a);
        hipFree(d_b);
        return err;
    }

    // Measure bandwidth
    hipEventRecord(start);
    err = hipMemcpy(d_b, d_a, size, hipMemcpyDeviceToDevice);
    if (err != hipSuccess) {
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_a);
        hipFree(d_b);
        return err;
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    // Calculate bandwidth in GB/s
    *bandwidth = (size / (1024.0f * 1024.0f * 1024.0f)) / (elapsed_time / 1000.0f);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_a);
    hipFree(d_b);

    return hipSuccess;
}
